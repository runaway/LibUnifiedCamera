#include "hip/hip_runtime.h"
#include <iostream>
#include<hip/hip_runtime.h>
#include<>
#include "DefectDetectionNoRotation.h"


#define shift 4


#define OFFSET_TIME 81

DefectDetectionNoRotation* DefectDetectionNoRotation::m_pInstance = NULL;		// ������ָ��
DefectDetectionNoRotation::DeleteItSelf DefectDetectionNoRotation::m_deleteInstance;


extern "C"
using namespace std;
using namespace cv;

__global__ static void bitwise_and_test(int width, int height, uchar* Images, uchar* immedian, int size_z, int* sum)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x >= size_z)
	{
		return;
	}
	int temp_temp_z = x / OFFSET_TIME;															//�ڼ���ͼ��
	int temp_z = x - (temp_temp_z * OFFSET_TIME);												//��temp_temp_z��ͼ���temp_z��zֵ
	int temp_y = temp_z / 9;															//yƫ����
	int temp_x = temp_z - ((temp_z / 9) * 9);											//xƫ����
	for (int temp_height = temp_y; temp_height < height - 8 + temp_y; temp_height++)	//˫forѭ�������������
	{
		for (int temp_width = temp_x; temp_width < width - 8 + temp_x; temp_width++)
		{
			int temp = abs(Images[temp_height*width + temp_width + (temp_temp_z)* width*height] - immedian[(temp_height - temp_y + 4)*width + (temp_width - temp_x + 4)]);
			sum[x] = sum[x] + temp;
		}
	}
	
}

DefectDetectionNoRotation::DefectDetectionNoRotation() :
	m_imgWid(0),
	m_imgHei(0),
	m_nNumImage(0),
	m_d_img_src(NULL),
	m_d_img_immedian(NULL),
	m_sum(NULL),
	m_sum1(NULL),
	m_offset_x(NULL),
	m_offset_y(NULL),
	m_sum_min(NULL),
	m_count(NULL)
{
}

DefectDetectionNoRotation::~DefectDetectionNoRotation()
{
	DeleteCUDAMemory();	
}

DefectDetectionNoRotation* DefectDetectionNoRotation::GetInstance()
{
	if (NULL == m_pInstance)
	{
		m_pInstance = new DefectDetectionNoRotation();
	}
	return m_pInstance;
}

void DefectDetectionNoRotation::image_abs(
    const Mat& immedian, 
    const cv::Rect& siteRect,
	vector<Mat>& images, 
	vector<Mat>& images_out)
{
	cout << "cuda no rotate" << endl;
	//��gpu�Ϸ���ռ�
	InitCUDAMemory(immedian.cols, immedian.rows, images.size());
	memset(m_sum1, 0, sizeof(int) * OFFSET_TIME * images.size());
	memset(m_offset_x, 0, sizeof(int) * images.size());
	memset(m_offset_y, 0, sizeof(int) * images.size());
	m_nNumImage = images.size();
	m_imgWid = immedian.cols;
	m_imgHei = immedian.rows;
	

	memset(m_count, 0, m_nNumImage * sizeof(int));
	//hipMemset(m_sum, 0, OFFSET_TIME * m_nNumImage * sizeof(int));
	hipMemcpy(m_d_img_immedian, immedian.data, m_imgWid * m_imgHei * sizeof(uchar), 
		hipMemcpyHostToDevice);
	hipMemcpy(m_sum, m_sum1, m_nNumImage * OFFSET_TIME * sizeof(int), hipMemcpyHostToDevice);
	dim3  grid(m_nNumImage * OFFSET_TIME / 32 + 1);
	dim3  block(32);
	for (int i = 0; i < m_nNumImage; i++)
	{
		hipMemcpy(&m_d_img_src[m_imgWid * m_imgHei * i], images[i].data,
			m_imgWid * m_imgHei * sizeof(uchar), hipMemcpyHostToDevice);
	}

	bitwise_and_test << <grid, block >> > (m_imgWid, m_imgHei, m_d_img_src, m_d_img_immedian,
		m_nNumImage * OFFSET_TIME, m_sum);

	hipMemcpy(m_sum1, m_sum, m_nNumImage * OFFSET_TIME * sizeof(int), hipMemcpyDeviceToHost);

	
	for (int i = 0; i < m_nNumImage; i++)
	{
		m_sum_min[i] = std::numeric_limits<int>::max();
	}	
	
	for (int m_temp = 0; m_temp < 81 * m_nNumImage; m_temp++)
	{
		int temp_z = m_temp / OFFSET_TIME;
		if (m_sum_min[temp_z] > m_sum1[m_temp])//sum_temp1����Ϊ�ռ���Сֵ�����飬m_temp_count����Ϊ�ռ�ƫ����������
		{
			m_count[temp_z] = m_temp - (m_temp / OFFSET_TIME) * OFFSET_TIME;
			m_sum_min[temp_z] = m_sum1[m_temp];
		}
	}

	for (int i = 0; i < m_nNumImage; i++)
	{
		m_offset_x[i] = (m_count[i]) - ((m_count[i] / 9) * 9) - 4;
		m_offset_y[i] = (m_count[i]) / 9 - 4;
	}
		

	for (int i = 0; i < m_nNumImage; i++)
	{
		Mat out = Mat::zeros(images[i].size(), images[i].type());
		images[i](Rect(shift + m_offset_x[i], shift + m_offset_y[i],images[i].cols - 2 * shift, 
			images[i].rows - 2 * shift)).copyTo(out(cv::Rect(shift, shift,images[i].cols - 2 * shift,
				images[i].rows - 2 * shift)));
		
		Mat diff = abs(immedian - out);
		
		Mat DiffRegions = Mat::zeros(diff.size(), diff.type());
		diff(siteRect).copyTo(DiffRegions(siteRect));
		
		images_out.push_back(DiffRegions);
	}	
}

void DefectDetectionNoRotation::ImageAbs(const Mat& immedian, const cv::Rect& siteRect,
	vector<Mat>& images, vector<Mat>& images_out)
{
	cout << "cuda no rotate" << endl;
	//��gpu�Ϸ���ռ�
	InitCUDAMemory(immedian.cols, immedian.rows, images.size());
	memset(m_sum1, 0, sizeof(int) * OFFSET_TIME * images.size());
	memset(m_offset_x, 0, sizeof(int) * images.size());
	memset(m_offset_y, 0, sizeof(int) * images.size());
	m_nNumImage = images.size();
	m_imgWid = immedian.cols;
	m_imgHei = immedian.rows;
	

	memset(m_count, 0, m_nNumImage * sizeof(int));
	//hipMemset(m_sum, 0, OFFSET_TIME * m_nNumImage * sizeof(int));
	hipMemcpy(m_d_img_immedian, immedian.data, m_imgWid * m_imgHei * sizeof(uchar), 
		hipMemcpyHostToDevice);
	hipMemcpy(m_sum, m_sum1, m_nNumImage * OFFSET_TIME * sizeof(int), hipMemcpyHostToDevice);
	dim3  grid(m_nNumImage * OFFSET_TIME / 32 + 1);
	dim3  block(32);
	for (int i = 0; i < m_nNumImage; i++)
	{
		hipMemcpy(&m_d_img_src[m_imgWid * m_imgHei * i], images[i].data,
			m_imgWid * m_imgHei * sizeof(uchar), hipMemcpyHostToDevice);
	}

	bitwise_and_test << <grid, block >> > (m_imgWid, m_imgHei, m_d_img_src, m_d_img_immedian,
		m_nNumImage * OFFSET_TIME, m_sum);

	hipMemcpy(m_sum1, m_sum, m_nNumImage * OFFSET_TIME * sizeof(int), hipMemcpyDeviceToHost);

	
	for (int i = 0; i < m_nNumImage; i++)
	{
		m_sum_min[i] = std::numeric_limits<int>::max();
	}	
	
	for (int m_temp = 0; m_temp < 81* m_nNumImage; m_temp++)
	{
		int temp_z = m_temp / OFFSET_TIME;
		if (m_sum_min[temp_z] > m_sum1[m_temp])//sum_temp1����Ϊ�ռ���Сֵ�����飬m_temp_count����Ϊ�ռ�ƫ����������
		{
			m_count[temp_z] = m_temp - (m_temp / OFFSET_TIME) * OFFSET_TIME;
			m_sum_min[temp_z] = m_sum1[m_temp];
		}
	}

	for (int i = 0; i < m_nNumImage; i++)
	{
		m_offset_x[i] = (m_count[i]) - ((m_count[i] / 9) * 9) - 4;
		m_offset_y[i] = (m_count[i]) / 9 - 4;
	}
		

	for (int i = 0; i < m_nNumImage; i++)
	{
		Mat out = Mat::zeros(images[i].size(), images[i].type());
		images[i](Rect(shift + m_offset_x[i], shift + m_offset_y[i],images[i].cols - 2 * shift, 
			images[i].rows - 2 * shift)).copyTo(out(cv::Rect(shift, shift,images[i].cols - 2 * shift,
				images[i].rows - 2 * shift)));
		
		Mat diff = abs(immedian - out);
		
		Mat DiffRegions = Mat::zeros(diff.size(), diff.type());
		diff(siteRect).copyTo(DiffRegions(siteRect));
		
		images_out.push_back(DiffRegions);
	}	
}

bool DefectDetectionNoRotation::InitCUDAMemory(int nWid, int nHei, int nNumImage)
{
	///lq ���Ѿ�����ǰ�������������ڴ棬���ٽ��з��䡣�ڴ�����ķ�ʱ��
	if (m_imgWid == nWid && m_imgHei == nHei && m_nNumImage == nNumImage
		&& NULL != m_sum && NULL != m_d_img_src && NULL != m_d_img_immedian)
	{
		return false;
	}

	//	cout << "InitCUDAMemory()!!!!!!!!!!!!!!!!!" << endl;

	///lq ��Ҫ���·��䣬���ͷ��ڴ�
	DeleteCUDAMemory();

	hipMalloc((void**)&m_d_img_immedian, nWid * nHei * sizeof(uchar));
	hipMalloc((void**)&m_d_img_src, nNumImage * nWid * nHei * sizeof(uchar));
	hipMalloc((void**)&m_sum, nNumImage * OFFSET_TIME * sizeof(int));


	m_sum1 = new int[OFFSET_TIME * nNumImage];			// ��Ӧsum
	m_offset_x = new int[nNumImage];						// ��Ӧƫ����x
	m_offset_y = new int[nNumImage];
	m_sum_min = new int[nNumImage];
	m_count = new int[nNumImage];

	return true;
}

bool DefectDetectionNoRotation::DeleteCUDAMemory()
{
	if (NULL != m_sum)
	{
		hipFree(m_sum);
		m_sum = NULL;
	}

	if (NULL != m_d_img_src)
	{
		hipFree(m_d_img_src);
		m_d_img_src = NULL;
	}

	if (NULL != m_d_img_immedian)
	{
		hipFree(m_d_img_immedian);
		m_d_img_immedian = NULL;
	}

	if (NULL != m_offset_x)
	{
		delete[] m_offset_x;
		m_offset_x = NULL;
	}

	if (NULL != m_offset_y)
	{
		delete[] m_offset_y;
		m_offset_y = NULL;
	}

	if (NULL != m_sum_min)
	{
		delete[] m_sum_min;
		m_sum_min = NULL;
	}

	if (NULL != m_count)
	{
		delete[] m_count;
		m_count = NULL;
	}

	return true;
}