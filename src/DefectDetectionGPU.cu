#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include<>
#include "DefectDetectionGPU.h"

#define shift 4

#define ROTATE_TIME  21
#define OFFSET_TIME 81
#define OPERATION_TIME  (21 * 81)
static float START_ANGLE = -1.0;
static float END_ANGLE = 1.0;
static float STEP_ANGLE = 0.1;
DefectDetectionGPU* DefectDetectionGPU::m_pInstance = NULL;		// ������ָ��
DefectDetectionGPU::DeleteItSelf DefectDetectionGPU::m_deleteInstance;


extern "C"
using namespace std;
using namespace cv;
 
//cuda�еĴ���ת��ģ��ƥ�����
__global__ static void rotate_abs(int width, int height, uchar* Images, uchar* immedian, int* sum, int size_t)

{ 
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x >= size_t)
	{
		return;
	}
	{
		int img_num = x / OPERATION_TIME;					//����ĵڼ���ͼ��
		int temp = x - img_num * OPERATION_TIME;			//��img_num��ͼ��ĵ�temp�δ�����Ϊ�����ֵ������
		int immedian_num = temp / OFFSET_TIME;				//��img_num��ͼ���Ӧ����ת�ĵ�immedian_num��ͼ��
		int temp_z = temp - OFFSET_TIME * immedian_num;		//��0��80��ƽ�Ƶĵ�temp_z�Σ���Ϊ��x,y��ƽ�ƾ��������
		int temp_y = temp_z / 9;							//y��ƽ�������ƶ��ľ���
		int temp_x = temp_z - temp_y * 9;					//x�������ƶ��ľ���
		for (int temp_height = temp_y; temp_height < height - 8 + temp_y; temp_height++)//˫forѭ�������������
		{
			for (int temp_width = temp_x; temp_width < width - 8 + temp_x; temp_width++)
			{
				int temp = abs(Images[temp_height*width + temp_width + img_num* width*height] - immedian[(temp_height - temp_y + 4)*width + (temp_width - temp_x + 4) + immedian_num*width*height]);
				sum[x] = sum[x] + temp;
			}
		}
	}
}

DefectDetectionGPU::DefectDetectionGPU():
	m_imgWid(0),
	m_imgHei(0),
	m_nNumImage(0),
	m_d_img_src(NULL),
	m_d_img_immedian_rotate(NULL),
	m_sum(NULL),
	m_sum1(NULL),
	m_offset_x(NULL),
	m_offset_y(NULL),
	m_offset_z(NULL),
	m_sum_min(NULL),
	m_count(NULL)
{
	
}

DefectDetectionGPU::~DefectDetectionGPU()
{
	DeleteCUDAMemory();	
}

DefectDetectionGPU* DefectDetectionGPU::GetInstance()
{
	if (NULL == m_pInstance)
	{
		m_pInstance = new DefectDetectionGPU();
	}
	return m_pInstance;
}

void DefectDetectionGPU::image_abs(const Mat& immedian1, const cv::Rect& siteRect, vector<Mat>& images,
	vector<Mat>& images_out)
{
	cout << "cuda with rotate" << endl;
	//��gpu�Ϸ���ռ�
	InitCUDAMemory(immedian1.cols, immedian1.rows, images.size());

	memset(m_sum1, 0, sizeof(int) * OPERATION_TIME * images.size());
	memset(m_offset_x, 0, sizeof(int) * images.size());
	memset(m_offset_y, 0, sizeof(int) * images.size());
	memset(m_offset_z, 0, sizeof(int) * images.size());
	
	m_nNumImage = images.size();
	m_imgWid = immedian1.cols;
	m_imgHei = immedian1.rows;

	for (int i = 0; i < m_nNumImage; i++)
	{
		m_sum_min[i] = std::numeric_limits<int>::max();
	}
	memset(m_count, 0, m_nNumImage * sizeof(int));
	hipMemset(m_sum, 0, OPERATION_TIME * m_nNumImage * sizeof(int));

	vector<Mat> rotate_images;	//images��Ϊ��ų�ʼͼ�����ݵ����飬rotate_images��Ϊ��ű�׼ͼ����ת��õ���ͼ������
	vector<float> float_temp;	//��Ŀ��ͼƬ������ת�ǶȲ�������float_temp��
	
	// ��¼��׼ͼ����תÿ���Ƕȵõ���ͼ��
	Mat image2;
	for (float j = START_ANGLE; j <= END_ANGLE; j += STEP_ANGLE)
	{
		image2 = immedian1.clone();
		float angle = j;
		Point2f center((float)(immedian1.cols / 2), (float)(immedian1.rows / 2));
		Mat affine_matrix = getRotationMatrix2D(center, angle, 1.0);//��ת����
		warpAffine(image2, image2, affine_matrix, immedian1.size());//��ת
		rotate_images.push_back(image2);//����ͼ������
	}
	
	//ʹ�����������Ϊgpu�ϵ����鸳��ֵ
	for (int i = 0; i < rotate_images.size(); i++)
	{
		hipMemcpy(&m_d_img_immedian_rotate[m_imgWid * m_imgHei * i],
			rotate_images[i].data, 
			m_imgWid * m_imgHei * sizeof(uchar), hipMemcpyHostToDevice);
	}

	for (int i = 0; i < m_nNumImage; i++)
	{
		hipMemcpy(&m_d_img_src[m_imgWid * m_imgHei * i], 
			images[i].data, m_imgWid * m_imgHei * sizeof(uchar),
			hipMemcpyHostToDevice);
	}

	//����cuda���߳���ͨ��gird��block�ķ��䣬����ʹ�õ���һά��
	dim3  grid(OPERATION_TIME * m_nNumImage / 32 + 1);
	dim3  block(32);
	rotate_abs <<<grid, block >>> (m_imgWid, m_imgHei, m_d_img_src,
		m_d_img_immedian_rotate, m_sum, OPERATION_TIME * m_nNumImage);

	//�����ݴ����ڴ�,����ֻ��������鴫��
	hipMemcpy(m_sum1, m_sum, OPERATION_TIME * m_nNumImage * sizeof(int), hipMemcpyDeviceToHost);

	//�������еĺ�ֵ��¼ÿ��ͼ���Ӧ��OPERATION_TIME����ֵ�е���Сֵ�Լ���Сֵ�Ķ�Ӧλ��
	for (int m_temp = 0; m_temp < OPERATION_TIME * m_nNumImage; m_temp++)
	{
		int temp_offset = m_temp / OPERATION_TIME;	//�ҳ�
		if (m_sum_min[temp_offset] > m_sum1[m_temp])
		{
			m_sum_min[temp_offset] = m_sum1[m_temp];
			m_count[temp_offset] = m_temp;
		}
	}

	//ͨ����Сֵ����Ӧ��λ�ã������ƫ����m_offset_x��m_offset_y����ת��m_offset_z
	for (int i = 0; i < m_nNumImage; i++)
	{
		int temp_offset = m_count[i] - OPERATION_TIME * i;
		m_offset_z[i] = temp_offset / OFFSET_TIME;
		int temp_xy_offset = temp_offset - (temp_offset / OFFSET_TIME) * OFFSET_TIME;
		m_offset_y[i] = temp_xy_offset / 9 - 4;
		m_offset_x[i] = temp_xy_offset - ((temp_xy_offset / 9) * 9) - 4;
	}

	//�洢������ͼ����
	for (int i = 0; i < m_nNumImage; i++)
	{
		Mat out = Mat::zeros(images[i].size(), images[i].type());
		images[i](Rect(shift + m_offset_x[i], shift + m_offset_y[i],
			images[i].cols - 2 * shift, images[i].rows 
			- 2 * shift)).copyTo(out(cv::Rect(shift, shift, 
				images[i].cols - 2 * shift, images[i].rows - 2 * shift)));

		Mat diff = abs(rotate_images[m_offset_z[i]] - out);

		Mat DiffRegions = Mat::zeros(diff.size(), diff.type());
		diff(siteRect).copyTo(DiffRegions(siteRect));

		//imwrite("image" + to_string(i) + ".bmp", diff);
		images_out.push_back(DiffRegions);
	}	
}

bool DefectDetectionGPU::InitCUDAMemory(int nWid, int nHei, int nNumImage)
{
	///lq ���Ѿ�����ǰ�������������ڴ棬���ٽ��з��䡣�ڴ�����ķ�ʱ��
	if (m_imgWid == nWid && m_imgHei == nHei && m_nNumImage == nNumImage
		&& NULL != m_sum && NULL != m_d_img_src && NULL != m_d_img_immedian_rotate)
	{
		return false;
	}

	cout << "InitCUDAMemory()!!!!!!!!!!!!!!!!!" << endl;

	///lq ��Ҫ���·��䣬���ͷ��ڴ�
	DeleteCUDAMemory();

	hipMalloc((void**)&m_d_img_immedian_rotate, ROTATE_TIME * nWid * nHei * sizeof(uchar));
	hipMalloc((void**)&m_d_img_src, nNumImage * nWid * nHei * sizeof(uchar));
	hipMalloc((void**)&m_sum, OPERATION_TIME * nNumImage * sizeof(int));	
	

	m_sum1 = new int[OPERATION_TIME * nNumImage];			// ��Ӧsum
	m_offset_x = new int[nNumImage];						// ��Ӧƫ����x
	m_offset_y = new int[nNumImage];						// ��Ӧƫ����y
	m_offset_z = new int[nNumImage];
	m_sum_min = new int[nNumImage];
	m_count = new int[nNumImage];
	

	return true;
}

bool DefectDetectionGPU::DeleteCUDAMemory()
{
	if (NULL != m_sum)
	{
		hipFree(m_sum);
		m_sum = NULL;
	}

	if (NULL != m_d_img_src)
	{
		hipFree(m_d_img_src);
		m_d_img_src = NULL;
	}

	if (NULL != m_d_img_immedian_rotate)
	{
		hipFree(m_d_img_immedian_rotate);
		m_d_img_immedian_rotate = NULL;
	}

	if (NULL != m_offset_x)
	{
		delete[] m_offset_x;
		m_offset_x = NULL;
	}

	if (NULL != m_offset_y)
	{
		delete[] m_offset_y;
		m_offset_y = NULL;
	}

	if (NULL != m_offset_z)
	{
		delete[] m_offset_z;
		m_offset_z = NULL;
	}

	if (NULL != m_sum_min)
	{
		delete[] m_sum_min;
		m_sum_min = NULL;
	}

	if (NULL != m_count)
	{
		delete[] m_count;
		m_count = NULL;
	}

	return true;
}